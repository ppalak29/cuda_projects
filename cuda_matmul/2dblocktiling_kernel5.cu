#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <random>

#define BM 128
#define BN 128
#define BK 8
#define TM 8
#define TN 8
#define MATRIX_SIZE 4096

__global__ void sgemm_2d_blocktiling(int M, int N, int K, float alpha, const float* A, const float* B, float beta, float* C) 
{
    const uint cRow = blockIdx.y;
    const uint cCol = blockIdx.x;
    
    const uint totalResultsBlocktile = BM * BN;
    const uint numThreadsBlocktile = totalResultsBlocktile / (TM * TN);
    
    __shared__ float As[BM * BK];
    __shared__ float Bs[BK * BN];
    
    const int threadCol = threadIdx.x % (BN / TN);
    const int threadRow = threadIdx.x / (BN / TN);
    
    const uint innerRowA = threadIdx.x / BK;
    const uint innerColA = threadIdx.x % BK;
    const uint strideA = numThreadsBlocktile / BK;
    
    const uint innerRowB = threadIdx.x / BN;
    const uint innerColB = threadIdx.x % BN;
    const uint strideB = numThreadsBlocktile / BN;
    
    A += cRow * BM * K;
    B += cCol * BN;
    C += cRow * BM * N + cCol * BN;
    
    float threadResults[TM * TN] = {0.0};
    float regM[TM] = {0.0};
    float regN[TN] = {0.0};
    
    for (uint bkIdx = 0; bkIdx < K; bkIdx += BK) {
        for (uint loadOffset = 0; loadOffset < BM; loadOffset += strideA) {
            As[(innerRowA + loadOffset) * BK + innerColA] = 
                A[(innerRowA + loadOffset) * K + innerColA];
        }
        
        for (uint loadOffset = 0; loadOffset < BK; loadOffset += strideB) {
            Bs[(innerRowB + loadOffset) * BN + innerColB] = 
                B[(innerRowB + loadOffset) * N + innerColB];
        }
        
        __syncthreads();
        
        A += BK;
        B += BK * N;
        
        for (uint dotIdx = 0; dotIdx < BK; ++dotIdx) {
            for (uint i = 0; i < TM; ++i) {
                regM[i] = As[(threadRow * TM + i) * BK + dotIdx];
            }
            
            for (uint i = 0; i < TN; ++i) {
                regN[i] = Bs[dotIdx * BN + threadCol * TN + i];
            }
            
            for (uint resIdxM = 0; resIdxM < TM; ++resIdxM) {
                for (uint resIdxN = 0; resIdxN < TN; ++resIdxN) {
                    threadResults[resIdxM * TN + resIdxN] += 
                        regM[resIdxM] * regN[resIdxN];
                }
            }
        }
        
        __syncthreads();
    }
    
    for (uint resIdxM = 0; resIdxM < TM; ++resIdxM) {
        for (uint resIdxN = 0; resIdxN < TN; ++resIdxN) {
            C[(threadRow * TM + resIdxM) * N + threadCol * TN + resIdxN] = 
                alpha * threadResults[resIdxM * TN + resIdxN] +
                beta * C[(threadRow * TM + resIdxM) * N + threadCol * TN + resIdxN];
        }
    }
}

void initialize_matrix(float* matrix, int n) {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dis(-1.0f, 1.0f);
    
    for (int i = 0; i < n * n; i++) {
        matrix[i] = dis(gen);
    }
}

int main() {
    const int n = MATRIX_SIZE;
    size_t size = n * n * sizeof(float);
    
    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);
    
    initialize_matrix(h_A, n);
    initialize_matrix(h_B, n);
    
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);
    
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice);
    
    dim3 gridDim((n + BN - 1) / BN, (n + BM - 1) / BM);
    dim3 blockDim((BM * BN) / (TM * TN));
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // Warmup
    sgemm_2d_blocktiling<<<gridDim, blockDim>>>(n, n, n, 1.0f, d_A, d_B, 0.0f, d_C);
    hipDeviceSynchronize();
    
    hipEventRecord(start);
    sgemm_2d_blocktiling<<<gridDim, blockDim>>>(n, n, n, 1.0f, d_A, d_B, 0.0f, d_C);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    long long flops = 2LL * n * n * n;
    double gflops = (flops / (milliseconds / 1000.0)) / 1e9;
    
    printf("\nKernel 5 Performance: \n");
    printf("Matrix size: %d x %d\n", n, n);
    printf("Duration: %.2f ms\n", milliseconds);
    printf("Performance: %.2f GFLOPs\n", gflops);
    printf("Percentage of peak (19.5 TF): %.2f%%\n", (gflops / 19500.0) * 100);
    
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    printf("C[0] = %f\n", h_C[0]);
    printf("Kernel completed successfully!\n");
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
    
    return 0;
}